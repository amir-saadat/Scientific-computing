// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// includes, project
#include <cutil.h>
#include <hipblas.h>
//=============================================================================
extern "C" void  cblas_saxpy(const int, const float, const float *, const int, 
                             const float *, const int);
extern "C" float cblas_snrm2(const int, const float *, const int);
extern "C" float cblas_isamax(const int, const float *, const int);
extern "C" void      sgeqrf_(int*,int*,float*,int*,float*,float*,int*,int*);
extern "C" int strmm_(char*, char *, char*, char *, int *, int *, float *,
                      float *, int *, float *, int *);
extern "C" int sgemm_(char *, char *, int *, int *, int *, float *, float *,
                      int *, float *, int *, float *, float *, int *);

void chol_qr_it(int m, int n, float *A, int lda, float *R);
void chol_qr_it_GPU(int m, int n, float *d_A, int lda, float *d_G, float *R, 
                 float *h_work, int lwork);
//=============================================================================

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    CUT_DEVICE_INIT(argc, argv);

    unsigned int timer = 0;

    /* Matrix size */
    int N, M;                 // NxM would be the size of the matrices 
                              // (M columns) that we would orthogonalize
    float *d_A, *d_G;         // d_A is array for A on the device (GPU)
    float *h_work, *h_tau;    // work space and array tau on the host
    float *h_A, *h_Q1, *h_Q2; // These would be the same NxM matrices 
    float *h_R, *h_G;

    int info[1], lwork, i;

    N  = 131072;
    M  = 128;    

    if (argc != 1)
    for(i = 1; i<argc; i++){	
      if (strcmp("-N", argv[i])==0)
         N = atoi(argv[++i]);
      else if (strcmp("-M", argv[i])==0)
         M = atoi(argv[++i]);
    }
    printf("\nUsage: \n");
    printf("  chol_qr_it -N %d -M %d\n\n", N, M);

    lwork = 2*N*M;

    int n2 = N * M;

    /* Initialize CUBLAS */
    cublasInit();

    /* Allocate host memory for the matrix */
    h_A  = (float*)malloc(n2 * sizeof( h_A[0]));
    h_Q1 = (float*)malloc(n2 * sizeof(h_Q1[0]));
    h_Q2 = (float*)malloc(n2 * sizeof(h_Q2[0]));
   
    h_G = (float*)malloc(M*M * sizeof(h_G[0]));
    h_R = (float*)malloc(M*M * sizeof(h_R[0]));
  
    CUDA_SAFE_CALL( hipHostMalloc( (void**)&h_work, lwork*4) );
  
    h_tau = (float*)malloc(N * sizeof(h_tau[0]));
   
    /* Take a random matrix h_A = h_Q1 = h_Q2 */
    for (i = 0; i < n2; i++) {
        h_A[i] = h_Q1[i] = h_Q2[i] = rand() / (float)RAND_MAX;
    }

    /* Allocate device memory for the matrices */
    cublasAlloc(n2, sizeof(d_A[0]), (void**)&d_A);
    cublasAlloc(M*M, sizeof(d_G[0]), (void**)&d_G);

    // create and start timer
    timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));

    /* =====================================================================
         Performs QR on CPU using LAPACK 
       ===================================================================== */
    sgeqrf_(&N, &M, h_A, &N, h_tau, h_work, &lwork, info);
    if (info[0] < 0)  
       printf("Argument %d of sgeqrf had an illegal value.\n", -info[0]);     

    // stop and destroy timer
    CUT_SAFE_CALL(cutStopTimer(timer));
    printf("CPU Processing time: %f (ms) \n", cutGetTimerValue(timer));
    printf("Speed: %f GFlops \n", 4.*N*M*M/
           (3.*1000000*cutGetTimerValue(timer)));
    CUT_SAFE_CALL(cutDeleteTimer(timer));


    /* Initialize the device matrix with the host matrices */
    hipblasSetVector(n2, sizeof(h_Q2[0]), h_Q2, 1, d_A, 1);

    timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));

    /* =====================================================================
         Performs orthogonalization on CPU using chol_qr_it
       ===================================================================== */
    chol_qr_it(N, M, h_Q2, N, h_R);

    // stop and destroy timer
    CUT_SAFE_CALL(cutStopTimer(timer));
    printf("\n\nCPU Processing time: %f (ms) \n", cutGetTimerValue(timer));
    printf("Speed: %f GFlops \n", 4.*N*M*M/
           (3.*1000000*cutGetTimerValue(timer)));
    CUT_SAFE_CALL(cutDeleteTimer(timer));

    float one = 1.f, zero = 0.f;
    sgemm_("t", "n", &M, &M, &N, &one, h_Q2, &N, h_Q2, &N, &zero, h_G, &M);
    for(i=0; i<M*M; i+=(M+1)) h_G[i] -= one;
    printf(" ||I - Q'Q||_F = %e, ||I-Q'Q||_max = %e \n",
            cblas_snrm2(M*M, h_G, 1), cblas_isamax(M*M, h_G, 1));

    strmm_("r", "u", "n", "n", &N, &M, &one, h_R, &M, h_Q2, &N);
    cblas_saxpy(n2, -1.0f, h_Q1, 1, h_Q2, 1);
    printf(" ||A - Q R||_F = %e \n",
            cblas_snrm2(n2, h_Q2, 1));    

    // chol_qr on GPU
    timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));

    /* =====================================================================
         Performs orthogonalization on CPU-GPU using chol_qr_it
       ===================================================================== */
    chol_qr_it_GPU(N, M, d_A, N, d_G, h_R, h_work, lwork);

    // stop and destroy timer
    CUT_SAFE_CALL(cutStopTimer(timer));
    printf("\n\nGPU Processing time: %f (ms) \n", cutGetTimerValue(timer));
    printf("Speed: %f GFlops \n", 4.*N*M*M/
           (3.*1000000*cutGetTimerValue(timer)));
    CUT_SAFE_CALL(cutDeleteTimer(timer));

    /* Read the result back */
    hipblasGetVector(n2, sizeof(h_Q2[0]), d_A, 1, h_Q2, 1);

    sgemm_("t", "n", &M, &M, &N, &one, h_Q2, &N, h_Q2, &N, &zero, h_G, &M);
    for(i=0; i<M*M; i+=(M+1)) h_G[i] -= one;
    printf(" ||I - Q'Q||_F = %e, ||I-Q'Q||_max = %e \n",
            cblas_snrm2(M*M, h_G, 1), cblas_isamax(M*M, h_G, 1));

    strmm_("r", "u", "n", "n", &N, &M, &one, h_R, &M, h_Q2, &N);
    cblas_saxpy(n2, -1.0f, h_Q1, 1, h_Q2, 1);
    printf(" ||A - Q R||_F = %e \n",
            cblas_snrm2(n2, h_Q2, 1));

    /* Memory clean up */
    free(h_A);
    free(h_Q1);
    free(h_Q2);
    free(h_R);
    free(h_G);
    CUDA_SAFE_CALL( hipHostFree(h_work) );
    free(h_tau);

    cublasFree(d_G);
    cublasFree(d_A);

    /* Shutdown */
    cublasShutdown();

    CUT_EXIT(argc, argv);
}
