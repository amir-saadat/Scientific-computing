#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cutil.h>
#include "hipblas.h"

//=============================================================================

extern "C" int sgemm_(char *, char *, int *, int *, int *, float *, float *, 
                      int *, float *, int *, float *, float *, int *);
extern "C" int sgesvd_(char *, char *, int *, int *, float *, int *, float *, 
                       float *, int *, float *, int *, float *, int *, int *);
extern "C" void sgeqrf_(int*, int*, float*, int*, float*, float*, int*, int*);
extern "C" int scopy_(int *, float*, int *, float*, int *);
extern "C" int strmm_(char*, char *, char*, char *, int *, int *, float *, 
                      float *, int *, float *, int *);
extern "C" int strsm_(char *, char *, char *, char *, int *, int *, 
                      float *, float *, int *, float *, int *);
extern "C" int ssyrk_(char *, char *, int *, int *, float *, float *, 
                      int *, float *, float *, int *);
//=============================================================================

void chol_qr_it(int m, int n, float *A, int lda, float *R){
    int i = 0, k, j, info, lwork = n*n, n2 = n*n, one = 1;
    float *G, *U, *S, *VT, *vt, *tau, *work;  
    float cn = 200.f, alpha = 1.f, zero = 0.f, mins, maxs;
    
    G    = (float*)malloc(n * n * 4);
    VT   = (float*)malloc(n * n * 4);
    S    = (float*)malloc(    n * 4);
    work = (float*)malloc(lwork * 4);
    tau  = (float*)malloc(    n * 4);  

    do {
      i++;

      sgemm_("t", "n", &n, &n, &m, &alpha, A, &m, A, &m, &zero, G, &n);
      //ssyrk_("l", "t", &n, &m, &alpha, A, &m, &zero, G, &n);
      //for(j=0; j<n; j++)
      //  for(k=0; k<j; k++)
      //     G[j*n+k] = G[k*n+j];

      sgesvd_("n", "a", &n, &n, G, &n, S, U, &n, VT, &n, work, &lwork, &info);

      mins = 100.f, maxs = 0.f;
      for(k=0; k<n; k++){
        S[k] = sqrt(S[k]);

	if (S[k] < mins)  mins = S[k];
	if (S[k] > maxs)  maxs = S[k];
      }

      for(k=0; k<n;k++){
        vt = VT + k*n;
        for(j=0; j<n; j++)
          vt[j]*=S[j];
      } 
      sgeqrf_(&n, &n, VT, &n, tau, work, &lwork, &info);

      if (i==1)
        scopy_(&n2, VT, &one, R, &one);
      else
        strmm_("l", "u", "n", "n", &n, &n, &alpha, VT, &n, R, &n);

      strsm_("r", "u", "n", "n", &m, &n, &alpha, VT, &n, A, &m);    

      if (mins > 0.00001f) 
        cn = maxs/mins;

      fprintf(stderr, "\nIteration %d, cond num = %f \n", i, cn);
    } while (cn > 100.f);

    free(G);
    free(VT);
    free(S);
    free(work);
    free(tau);
}

//=============================================================================

void chol_qr_it_GPU(int m, int n, float *d_A, int lda, float *G, float *R, 
                 float *work, int lwork){
    int i = 0, k, j, info, n2 = n*n, one = 1, lwork2 = n*n;
    float *U, *S, *VT, *vt, *tau, *work1, *work2;
    float cn = 200.f, alpha = 1.f, zero = 0.f, mins, maxs;

    work1 = (float*) malloc(n * n * 4); 
	VT   = (float*)malloc(n * n * 4);
    work2 = (float*)malloc(lwork2 * 4);
    S    = (float*)malloc(    n * 4);
    tau  = (float*)malloc(    n * 4);

    do {
      i++;
      hipblasSgemm('T', 'N', n, n, m, alpha, d_A, lda, d_A, lda, zero, G, n);
      hipblasGetVector(n2 , 4.f ,G  , 1.f , work1 , 1.f);

      sgesvd_("N", "A", &n, &n, work1, &n, S, U, &n, VT, &n, work2, &lwork2, &info);

      mins = 100.f, maxs = 0.f;
      for(k=0; k<n; k++){
        S[k] = sqrt(S[k]);

        if (S[k] < mins)  mins = S[k];
        if (S[k] > maxs)  maxs = S[k];
      }

      for(k=0; k<n;k++){
        vt = VT + k*n;
        for(j=0; j<n; j++)
          vt[j]*=S[j];
      }
      sgeqrf_(&n, &n, VT, &n, tau, work2, &lwork2, &info);

      if (i==1)
        scopy_(&n2, VT, &one, R, &one);/*then for next loops, R keeps staying upper triangular*/
      else
        strmm_("l", "u", "n", "n", &n, &n, &alpha, VT, &n, R, &n);/*R is "u" from pre loop, so thing are right*/

      hipblasSetVector(n2, 4.f, VT, 1.f, G, 1.f);
      hipblasStrsm('r', 'u', 'n', 'n', m, n, alpha, G, n, d_A, m);

      if (mins > 0.00001f)
        cn = maxs/mins;
      
      fprintf(stderr, "\nIteration %d, cond num = %f \n", i, cn);
    } while (cn > 100.f);

    free(VT);
    free(S);
    free(tau);
	free(work1);
	free(work2);
}

//=============================================================================
